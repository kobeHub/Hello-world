
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>


// CUDA kernel function using GPU
__global__
void add(int n, float *a, float *b) {
  int index = threadIdx.x;
  int stride = blockDim.x;
  
  for (int i = index; i < n; i += stride) {
    b[i] += a[i];
  }
}

int main() {
  const int N = 1 << 20;
  float *a, *b;
  hipMallocManaged(&a, N*sizeof(float));
  hipMallocManaged(&b, N*sizeof(float));

  // Init arrays from host
  for (int i = 0; i < N; i++) {
    a[i] = 0.1f;
    b[i] = 0.2f;
  }

  // Run kernel function using tripe angle bracket
  add<<<1, 256>>>(N, a, b);

  // Wait for GPU results before hosts process
  hipDeviceSynchronize();

  // Check errors
  // all elements of b should be 0.3f
  float max_error = 0.0f;
  for (int i = 0; i < N; i++) {
    max_error = fmax(max_error, fabs(b[i] - 0.3f));
  }
  std::cout << "Max error using GPU: " << max_error << std::endl;

  hipFree(a);
  hipFree(b);
}
